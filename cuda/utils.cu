#include "utils.cuh"

TimingGPU::TimingGPU() { timer = std::make_unique<TimerGPU>(); }

void TimingGPU::StartCounter()
{
    hipEventCreate(&((*timer).start));
    hipEventCreate(&((*timer).stop));
    hipEventRecord((*timer).start,0);
}

void TimingGPU::StartCounterFlags()
{
    int eventflags = hipEventBlockingSync;

    hipEventCreateWithFlags(&((*timer).start),eventflags);
    hipEventCreateWithFlags(&((*timer).stop),eventflags);
    hipEventRecord((*timer).start,0);
}

// Gets the counter in ms
float TimingGPU::GetCounter()
{
    float time;
    hipEventRecord((*timer).stop, 0);
    hipEventSynchronize((*timer).stop);
    hipEventElapsedTime(&time,(*timer).start,(*timer).stop);
    return time;
}
