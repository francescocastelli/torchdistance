#include "hip/hip_runtime.h"
#include <torch/extension.h> 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace {

template <typename scalar_t>
__global__ void distance_cuda_kernel(
    scalar_t* const __restrict__ src, 
    scalar_t* const __restrict__ trg, 
    int* __restrict__ result,
    int* dMatrix,
    int64_t srcLen,
    int64_t trgLen, 
    int64_t padToken) {
    
    const int batch = blockIdx.x;
    int cols = trgLen+1;

    auto srcBatch = src + batch * srcLen;
    auto trgBatch = trg + batch * trgLen;
    auto result_ = result + batch;
    auto d = dMatrix + (batch * (trgLen+1) * 2);

    // handle padding
    for (int i=0; i < srcLen; i++)
    {
	    if (srcBatch[i] == padToken)
	    {
		    srcLen = i;
		    break;
	    }
    }

    for (int i=0; i < trgLen; i++)
    {
	    if (trgBatch[i] == padToken)
	    {
		    trgLen = i;
		    break;
	    }
    }

    // one or both strings are null
    if (srcLen == 0) 
    {
	    *result = trgLen; 
	    return;
    }
    else if (trgLen == 0) 
    {
	    *result = srcLen; 
	    return;
    }

    auto src_ = srcBatch, trg_ = trgBatch;
    auto srcLen_ = srcLen, trgLen_ = trgLen;
    if (trgLen < srcLen) src_ = trgBatch, trg_ = srcBatch, srcLen_ = trgLen, trgLen_ = srcLen;

    d[0] = 0;
    d[cols] = 1;
    for (int i = 0; i < trgLen_ + 1; i++) d[i] = i;
    for (int i = 1; i < srcLen_ + 1; i++) {
        for (int j = 1; j < trgLen_ + 1; j++) {
            d[(i&1)*cols + j] = std::min(std::min(d[((i-1)&1)*cols + j], d[(i&1)*cols + (j-1)]) + 1, 
			    		 d[((i-1)&1)*cols + (j-1)] + (src_[i-1] == trg_[j-1] ? 0 : 1));
        }
    }

    *result_ = d[(srcLen_&1)*cols + trgLen_];
}
}

torch::Tensor editdistance_cuda_kernel(
    const torch::Tensor& src, 
    const torch::Tensor& trg, 
    torch::Tensor& result, 
    int64_t padToken) {

    const auto numBatch = src.size(0);
    const auto srcLen = src.size(1);
    const auto trgLen = trg.size(1);

    const int threads = 1;
    const dim3 blocks(numBatch);

    int* d;
    hipMalloc(&d, numBatch * 2 * (trgLen+1) * sizeof(int));

    AT_DISPATCH_ALL_TYPES(
        src.scalar_type(),
        "editdistance_cuda",
        ([&] {
         distance_cuda_kernel<scalar_t><<<numBatch, threads>>>(
            src.data<scalar_t>(),
            trg.data<scalar_t>(),
            result.data<int>(),
	    d,
            srcLen, 
            trgLen, 
	    padToken);
        }));

    hipFree(&d);
    return result;
}
